
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

__global__ void guard(float *A) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init(23497214, tid, 0, &state);
    printf("guard kernel starts\n");
    float out = 0;
    while (true) {
        int id = hiprand(&state) % (3 << 19);
        out += A[id];
        if (id == 0) break;
    }
    A[0] = out;
}

int main(void)
{
    int numElements = 3 << 19;
    size_t size = numElements * sizeof(float);
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    // Launch the guard Kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = 1;
    guard<<<blocksPerGrid, threadsPerBlock>>>(d_A);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    return 0;
}

