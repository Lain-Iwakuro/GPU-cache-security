
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#define N 1
#include <fstream>
#include <sstream>
#include <vector>

#define ROWS 32
#define COLS 16
// #define LDCG 1
#define REPEAT 50
#define REPEAT_IN 5

__global__ void monitor(float *A, int* M, float *C) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState state;
    hiprand_init(46346232, tid, 0, &state);
    clock_t start, end;

    if (tid > 0) return;

    int line = 2;

    int idx0 = M[line * 16 + 0];
    int idx1 = M[line * 16 + 1];
    int idx2 = M[line * 16 + 2];
    int idx3 = M[line * 16 + 3];
    int idx4 = M[line * 16 + 4];
    int idx5 = M[line * 16 + 5];
    int idx6 = M[line * 16 + 6];
    int idx7 = M[line * 16 + 7];
    int idx8 = M[line * 16 + 8];
    int idx9 = M[line * 16 + 9];
    int idx10 = M[line * 16 + 10];
    int idx11 = M[line * 16 + 11];
    int idx12 = M[line * 16 + 12];
    int idx13 = M[line * 16 + 13];
    int idx14 = M[line * 16 + 14];
    int idx15 = M[line * 16 + 15];

    float test = 0;

    // warm up
    float warm = 0;
    int size = 3 << 19, float_per_line = 16;
    if (tid == 0) {
        for (int k = 0; k < 3; k++) {
            for (int idx = 0; idx < size; idx += float_per_line) {
                warm += A[idx];
            }
        }
    }
    
    float out = 0;

#pragma unroll
    for (int k = 0; k < REPEAT; k++) {
        // put into cache
#pragma unroll
        for (int i = 0; i < REPEAT_IN; i++) {
            start = clock64();
            out += C[idx0];
            out += C[idx1];
            out += C[idx2];
            out += C[idx3];
            out += C[idx4];
            out += C[idx5];
            out += C[idx6];
            out += C[idx7];
            out += C[idx8];
            out += C[idx9];
            out += C[idx10];
            out += C[idx11];
            out += C[idx12];
            out += C[idx13];
            out += C[idx14];
            out += C[idx15];
            end = clock64();
            // printf("%f\n", (float)(end - start));
        }
        // __syncthreads();

        // access
        out += A[57696 - line * 32];
        // __syncthreads();


        // check
#pragma unroll
        for (int i = 0; i < REPEAT_IN; i++) {
            start = clock64();
            out += C[idx0];
            out += C[idx1];
            out += C[idx2];
            out += C[idx3];
            out += C[idx4];
            out += C[idx5];
            out += C[idx6];
            out += C[idx7];
            out += C[idx8];
            out += C[idx9];
            out += C[idx10];
            out += C[idx11];
            out += C[idx12];
            out += C[idx13];
            out += C[idx14];
            out += C[idx15];
            end = clock64();
            printf("%f\n", (float)(end - start));
        }
    }

    A[0] = out;
    A[1] = warm;
    A[2] = test;
}



int main() {
    std::ifstream file("../rev_set.txt");
    if (!file.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return EXIT_FAILURE;
    }

    // int array[ROWS][COLS];
    int *h_M = (int *)malloc(ROWS * COLS * sizeof(int));
    std::string line;
    for (int i = 0; i < ROWS && std::getline(file, line); ++i) {
        std::istringstream iss(line);
        std::string number;
        for (int j = 0; j < COLS && std::getline(iss, number, ','); ++j) {
            // array[i][j] = std::stoi(number);
            h_M[i * COLS + j] = std::stoi(number);
        }
    }

    file.close();

    // Print the array to verify
    // for (int i = 0; i < ROWS; ++i) {
    //     for (int j = 0; j < COLS; ++j) {
    //         // std::cout << array[i][j] << " ";
    //         std::cout << h_M[i * COLS + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    int numElements = 3 << 19;
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    // Allocate the device output vector C
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    int *d_M = NULL;
    hipMalloc((void **)&d_M, ROWS * COLS * sizeof(int));

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, ROWS * COLS * sizeof(int), hipMemcpyHostToDevice);

    // Launch the Kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = 1;
    monitor<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_M, d_C);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_C);

    // printf("Done\n");
    return 0;
}
