
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

#define PRIME_REP 2
#define ACCESS_REP 2
#define CHECK_REP 5

// use this kernel to figure out which addresses map to the same set
__global__ void get_set(const float *A,  float *C, int numElements)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid > 0) return;
    // printf("hello, world!\n");
    hiprandState state;
    hiprand_init(136782492, gid, 0, &state);
    clock_t start, end;

    int size = 3 << 19, float_per_line = 16;
    // int idx_start = 3 << 19;
    int idx_start = 0;
    int send_stride = 32;
    float out = 0, time;
    int candidate[CHECK_REP];
    int last_idx = 0;

    // warm_up
    float warm = 0;
    for (int k = 0; k < PRIME_REP; k++) {
        for (int idx = 0; idx < size; idx += float_per_line) {
            warm += A[idx];
        }
    }

    for (int send_idx = idx_start; send_idx < idx_start + size; send_idx += send_stride) {
#pragma unroll
        for (int k = 0; k < CHECK_REP; k++) {
            // printf("write A to L2 cache!\n");
#pragma unroll
            for (int j = 0; j < PRIME_REP; j++) {
                for (int idx = 0; idx < size; idx += float_per_line) {
                    out += A[idx];
                }
            }

            // printf("fit cache line at send_idx into cache!\n");
#pragma unroll
            for (int j = 0; j < ACCESS_REP; j++) {
                out += C[send_idx];
            }

            // printf("get the cache line that is gone!\n");
            for (int offset = 0; offset < size; offset += send_stride) {
                int recv_idx = (last_idx + offset) % size;
                start = clock64();
                out += A[recv_idx];
                end = clock64();
                time = (float)(end - start);
                // printf("%f\n", time);
                if (time > 350.0) {
                    // printf("%d, %f\n", recv_idx, time);
                    candidate[k] = recv_idx;
                    // last_idx = (recv_idx + size - 1024) % size;
                    last_idx = 0;
                    // printf("%d\n", candidate[k]);
                    break;
                }
            }
        }
        // verify
        printf("%d", send_idx);
        for (int k = 0; k < CHECK_REP - 1; k++) {
            if (candidate[k] == candidate[k + 1]) {
                printf(", %d", candidate[k]);
                break;
            }
        }
        printf("\n");
    }
    C[0] = out;
}

int main(void)
{
    int numElements = 3 << 19;
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand() / (float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    // Allocate the device output vector C
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    // Copy the host input
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // Launch the CUDA Kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = 1;
    get_set<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, numElements);

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_C);

    // printf("Done\n");
    return 0;
}